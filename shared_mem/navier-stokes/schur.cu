#include <iostream>
#include <string>

#include <amgcl/adapter/crs_tuple.hpp>
#include <amgcl/backend/cuda.hpp>
#include <amgcl/make_solver.hpp>
#include <amgcl/preconditioner/schur_pressure_correction.hpp>
#include <amgcl/solver/fgmres.hpp>
#include <amgcl/solver/bicgstab.hpp>
#include <amgcl/coarsening/smoothed_aggregation.hpp>
#include <amgcl/relaxation/as_preconditioner.hpp>
#include <amgcl/relaxation/damped_jacobi.hpp>
#include <amgcl/relaxation/ilu0.hpp>
#include <amgcl/amg.hpp>
#include <amgcl/io/binary.hpp>
#include <amgcl/profiler.hpp>

#include "log_times.hpp"
#include "argh.h"
    
namespace amgcl { profiler<amgcl::backend::cuda_clock> prof; }
using amgcl::prof;

//---------------------------------------------------------------------------
int main(int argc, char *argv[]) {
    using namespace amgcl;

    argh::parser cmdl(argc, argv);

    double tol;
    cmdl({"e", "tol"}, "1e-4") >> tol;

    size_t rows, n, m;
    std::vector<ptrdiff_t> ptr, col;
    std::vector<double> val, rhs;

    prof.tic("reading");
    io::read_crs(cmdl({"A", "matrix"}, "A.bin").str(), rows, ptr, col, val);
    io::read_dense(cmdl({"f", "rhs"}, "b.bin").str(), n, m, rhs);
    prof.toc("reading");

    typedef backend::cuda<double> Backend;
    typedef make_solver<
        preconditioner::schur_pressure_correction<
            make_solver<
                relaxation::as_preconditioner<Backend, relaxation::damped_jacobi>,
                solver::bicgstab<Backend>
            >,
            make_solver<
                amg<
                    Backend,
                    coarsening::smoothed_aggregation,
                    relaxation::ilu0
                    >,
                solver::fgmres<Backend>
                >
            >,
        solver::fgmres<Backend>
        > Solver;

    Backend::params bprm;
    hipsparseCreate(&bprm.cusparse_handle);

    Solver::params prm;

    prm.solver.tol = tol;
    prm.precond.usolver.solver.tol = tol * 10;
    prm.precond.psolver.solver.tol = tol * 10;

    cmdl("pressure-iters", "16") >> prm.precond.psolver.solver.maxiter;
    prm.precond.psolver.precond.relax.solve.iters = 3;

    prm.precond.pmask.resize(n, 0);
    for(size_t i = 0; i < rows; i += 4)
        prm.precond.pmask[i] = 1;


    prof.tic("setup");
    Solver solve(std::tie(rows, ptr, col, val), prm, bprm);
    double tm_setup = prof.toc("setup");

    std::cout << solve << std::endl;

    thrust::device_vector<double> f = rhs;
    thrust::device_vector<double> x(n, 0.0);

    int iters;
    double error;

    prof.tic("solve");
    std::tie(iters, error) = solve(f, x);
    double tm_solve = prof.toc("solve");

    std::cout
        << "iters: " << iters << std::endl
        << "error: " << error << std::endl
        << prof << std::endl;

    log_times("amgcl-schur-cuda.txt", 1, rows, iters, tm_setup, tm_solve);
}
