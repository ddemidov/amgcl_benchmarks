#include <iostream>
#include <vector>

#include <boost/property_tree/ptree.hpp>

#include <amgcl/value_type/static_matrix.hpp>
#include <amgcl/backend/cuda.hpp>
#include <amgcl/adapter/crs_tuple.hpp>
#include <amgcl/adapter/block_matrix.hpp>
#include <amgcl/make_solver.hpp>
#include <amgcl/amg.hpp>
#include <amgcl/coarsening/aggregation.hpp>
#include <amgcl/relaxation/damped_jacobi.hpp>
#include <amgcl/solver/lgmres.hpp>
#include <amgcl/io/binary.hpp>
#include <amgcl/profiler.hpp>

#include "log_times.hpp"
#include "argh.h"

namespace amgcl { profiler<backend::cuda_clock> prof; }
using amgcl::prof;

//---------------------------------------------------------------------------
int main(int argc, char *argv[]) {
    using namespace amgcl;

    typedef backend::cuda<double> Backend;
    typedef make_solver<
        amg<Backend, coarsening::aggregation, relaxation::damped_jacobi>,
        solver::lgmres<Backend>
        > Solver;

    argh::parser cmdl(argc, argv);

    Backend::params bprm;
    hipsparseCreate(&bprm.cusparse_handle);

    Solver::params prm;
    prm.solver.maxiter = 500;
    cmdl({"e", "tol"}, "1e-4") >> prm.solver.tol;
    prm.precond.coarsening.aggr.block_size = 4;

    size_t rows, n, m;
    std::vector<ptrdiff_t> ptr, col;
    std::vector<double> val, f;

    prof.tic("read");
    io::read_crs(cmdl({"A", "matrix"}, "A.bin").str(), rows, ptr, col, val);
    io::read_dense(cmdl({"f", "rhs"}, "b.bin").str(), n, m, f);
    prof.toc("read");

    assert(n == rows && m == 1);

    prof.tic("setup");
    Solver solve(std::tie(rows, ptr, col, val), prm, bprm);
    double tm_setup = prof.toc("setup");

    std::cout << solve << std::endl;

    thrust::device_vector<double> F = f;
    thrust::device_vector<double> X(n, 0.0);

    int iters;
    double error;

    prof.tic("solve");
    std::tie(iters, error) = solve(F, X);
    double tm_solve = prof.toc("solve");

    std::cout
        << "iters: " << iters << std::endl
        << "error: " << error << std::endl
        << prof << std::endl;

    log_times("amgcl-cuda.txt", 1, rows, iters, tm_setup, tm_solve);
}
