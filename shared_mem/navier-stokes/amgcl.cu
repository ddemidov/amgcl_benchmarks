#include <iostream>
#include <vector>

#include <boost/program_options.hpp>
#include <boost/property_tree/ptree.hpp>

#include <amgcl/value_type/static_matrix.hpp>
#include <amgcl/backend/cuda.hpp>
#include <amgcl/adapter/crs_tuple.hpp>
#include <amgcl/adapter/block_matrix.hpp>
#include <amgcl/make_solver.hpp>
#include <amgcl/amg.hpp>
#include <amgcl/coarsening/aggregation.hpp>
#include <amgcl/relaxation/damped_jacobi.hpp>
#include <amgcl/solver/lgmres.hpp>
#include <amgcl/io/binary.hpp>
#include <amgcl/profiler.hpp>

#include "log_times.hpp"

namespace amgcl { profiler<backend::cuda_clock> prof; }
using amgcl::prof;

//---------------------------------------------------------------------------
int main(int argc, char *argv[]) {
    using namespace amgcl;
    namespace po = boost::program_options;

    typedef backend::cuda<double> Backend;
    typedef make_solver<
        amg<Backend, coarsening::aggregation, relaxation::damped_jacobi>,
        solver::lgmres<Backend>
        > Solver;

    po::options_description desc("Options");

    desc.add_options()
        ("help,h", "Show this help.")
        ("matrix,A",
         po::value<std::string>()->default_value("A.bin"),
         "System matrix in binary format."
        )
        (
         "rhs,f",
         po::value<std::string>()->default_value("b.bin"),
         "The RHS vector in binary format."
        )
        (
         "tol,e",
         po::value<double>()->default_value(1e-4),
         "Tolerance"
        )
        ;

    po::variables_map vm;
    po::store(po::parse_command_line(argc, argv, desc), vm);
    po::notify(vm);

    if (vm.count("help")) {
        std::cout << desc << std::endl;
        return 0;
    }

    Backend::params bprm;
    hipsparseCreate(&bprm.cusparse_handle);

    Solver::params prm;
    prm.solver.maxiter = 500;
    prm.solver.tol = vm["tol"].as<double>();
    prm.precond.coarsening.aggr.block_size = 4;

    size_t rows, n, m;
    std::vector<ptrdiff_t> ptr, col;
    std::vector<double> val, f;

    prof.tic("read");
    io::read_crs(vm["matrix"].as<std::string>(), rows, ptr, col, val);
    io::read_dense(vm["rhs"].as<std::string>(), n, m, f);
    prof.toc("read");

    assert(n == rows && m == 1);

    prof.tic("setup");
    Solver solve(boost::tie(rows, ptr, col, val), prm, bprm);
    double tm_setup = prof.toc("setup");

    std::cout << solve << std::endl;

    thrust::device_vector<double> F = f;
    thrust::device_vector<double> X(n, 0.0);

    int iters;
    double error;

    prof.tic("solve");
    boost::tie(iters, error) = solve(F, X);
    double tm_solve = prof.toc("solve");

    std::cout
        << "iters: " << iters << std::endl
        << "error: " << error << std::endl
        << prof << std::endl;

    log_times("amgcl-cuda.txt", 1, rows, iters, tm_setup, tm_solve);
}
