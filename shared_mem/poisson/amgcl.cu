#include <iostream>
#include <vector>

#include <boost/program_options.hpp>

#include <amgcl/backend/cuda.hpp>
#include <amgcl/adapter/crs_tuple.hpp>
#include <amgcl/make_solver.hpp>
#include <amgcl/amg.hpp>
#include <amgcl/coarsening/smoothed_aggregation.hpp>
#include <amgcl/relaxation/spai0.hpp>
#include <amgcl/solver/cg.hpp>
#include <amgcl/profiler.hpp>

#include "log_times.hpp"

//---------------------------------------------------------------------------
void assemble(
        int n,
        std::vector<int>    &ptr,
        std::vector<int>    &col,
        std::vector<double> &val
        )
{
    int n3 = n * n * n;

    ptr.clear(); ptr.reserve(n3 + 1);
    col.clear(); col.reserve(n3 * 7);
    val.clear(); val.reserve(n3 * 7);

    ptr.push_back(0);

    for(int k = 0, idx = 0; k < n; ++k) {
        for(int j = 0; j < n; ++j) {
            for (int i = 0; i < n; ++i, ++idx) {
                if (k > 0) {
                    col.push_back(idx - n * n);
                    val.push_back(-1.0/6.0);
                }

                if (j > 0) {
                    col.push_back(idx - n);
                    val.push_back(-1.0/6.0);
                }

                if (i > 0) {
                    col.push_back(idx - 1);
                    val.push_back(-1.0/6.0);
                }

                col.push_back(idx);
                val.push_back(1.0);

                if (i + 1 < n) {
                    col.push_back(idx + 1);
                    val.push_back(-1.0/6.0);
                }

                if (j + 1 < n) {
                    col.push_back(idx + n);
                    val.push_back(-1.0/6.0);
                }

                if (k + 1 < n) {
                    col.push_back(idx + n * n);
                    val.push_back(-1.0/6.0);
                }

                ptr.push_back(col.size());
            }
        }
    }
}

//---------------------------------------------------------------------------
int main(int argc, char *argv[]) {
    using namespace amgcl;
    namespace po = boost::program_options;

    po::options_description desc("Options");

    desc.add_options()
        ("help,h", "Show this help.")
        (
         "size,n",
         po::value<int>()->default_value(150),
         "The size of the Poisson problem to solve when no system matrix is given. "
         "Specified as number of grid nodes along each dimension of a unit cube. "
         "The resulting system will have n*n*n unknowns. "
        )
        ;

    po::variables_map vm;
    po::store(po::parse_command_line(argc, argv, desc), vm);
    po::notify(vm);

    if (vm.count("help")) {
        std::cout << desc << std::endl;
        return 0;
    }

    typedef
        backend::cuda<double> Backend;

    typedef
        make_solver<
            amg<
                Backend,
                coarsening::smoothed_aggregation,
                relaxation::spai0
                >,
            solver::cg<Backend>
        > Solver;

    Backend::params bprm;
    hipsparseCreate(&bprm.cusparse_handle);

    Solver::params prm;
    prm.precond.coarsening.relax = 0.75;

    const int n = vm["size"].as<int>();
    const int n3 = n * n * n;

    std::vector<int> ptr, col;
    std::vector<double> val;

    profiler<backend::cuda_clock> prof;

    prof.tic("assemble");
    assemble(n, ptr, col, val);
    prof.toc("assemble");

    thrust::device_vector<double> f(n3, 1.0);
    thrust::device_vector<double> x(n3, 0.0);

    prof.tic("setup");
    Solver solve(boost::tie(n3, ptr, col, val), prm, bprm);
    double tm_setup = prof.toc("setup");

    std::cout << solve << std::endl;

    int iters;
    double error;

    prof.tic("solve");
    boost::tie(iters, error) = solve(f, x);
    double tm_solve = prof.toc("solve");

    std::cout
        << "iters: " << iters << std::endl
        << "error: " << error << std::endl
        << prof << std::endl;

    log_times("amgcl-cuda.txt", 1, n, iters, tm_setup, tm_solve);
}
