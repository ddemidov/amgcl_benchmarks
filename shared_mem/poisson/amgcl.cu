#include <iostream>
#include <vector>

#include <amgcl/backend/cuda.hpp>
#include <amgcl/adapter/crs_tuple.hpp>
#include <amgcl/make_solver.hpp>
#include <amgcl/amg.hpp>
#include <amgcl/coarsening/smoothed_aggregation.hpp>
#include <amgcl/relaxation/spai0.hpp>
#include <amgcl/solver/cg.hpp>
#include <amgcl/profiler.hpp>

#include "log_times.hpp"
#include "argh.h"

namespace amgcl { profiler<backend::cuda_clock> prof; }
using amgcl::prof;

//---------------------------------------------------------------------------
void assemble(
        int n,
        std::vector<int>    &ptr,
        std::vector<int>    &col,
        std::vector<double> &val
        )
{
    int n3 = n * n * n;

    ptr.clear(); ptr.reserve(n3 + 1);
    col.clear(); col.reserve(n3 * 7);
    val.clear(); val.reserve(n3 * 7);

    ptr.push_back(0);

    for(int k = 0, idx = 0; k < n; ++k) {
        for(int j = 0; j < n; ++j) {
            for (int i = 0; i < n; ++i, ++idx) {
                if (k > 0) {
                    col.push_back(idx - n * n);
                    val.push_back(-1.0/6.0);
                }

                if (j > 0) {
                    col.push_back(idx - n);
                    val.push_back(-1.0/6.0);
                }

                if (i > 0) {
                    col.push_back(idx - 1);
                    val.push_back(-1.0/6.0);
                }

                col.push_back(idx);
                val.push_back(1.0);

                if (i + 1 < n) {
                    col.push_back(idx + 1);
                    val.push_back(-1.0/6.0);
                }

                if (j + 1 < n) {
                    col.push_back(idx + n);
                    val.push_back(-1.0/6.0);
                }

                if (k + 1 < n) {
                    col.push_back(idx + n * n);
                    val.push_back(-1.0/6.0);
                }

                ptr.push_back(col.size());
            }
        }
    }
}

//---------------------------------------------------------------------------
int main(int argc, char *argv[]) {
    using namespace amgcl;

    typedef
        backend::cuda<double> Backend;

    typedef
        make_solver<
            amg<
                Backend,
                coarsening::smoothed_aggregation,
                relaxation::spai0
                >,
            solver::cg<Backend>
        > Solver;

    Backend::params bprm;
    hipsparseCreate(&bprm.cusparse_handle);

    Solver::params prm;
    prm.precond.coarsening.relax = 0.75;

    int n;

    argh::parser cmdl(argc, argv);
    cmdl({"n", "size"}, "150") >> n;
    int n3 = n * n * n;

    std::vector<int> ptr, col;
    std::vector<double> val;

    prof.tic("assemble");
    assemble(n, ptr, col, val);
    prof.toc("assemble");

    thrust::device_vector<double> f(n3, 1.0);
    thrust::device_vector<double> x(n3, 0.0);

    prof.tic("setup");
    Solver solve(std::tie(n3, ptr, col, val), prm, bprm);
    double tm_setup = prof.toc("setup");

    std::cout << solve << std::endl;

    int iters;
    double error;

    prof.tic("solve");
    std::tie(iters, error) = solve(f, x);
    double tm_solve = prof.toc("solve");

    std::cout
        << "iters: " << iters << std::endl
        << "error: " << error << std::endl
        << prof << std::endl;

    log_times("amgcl-cuda.txt", 1, n, iters, tm_setup, tm_solve);
}
